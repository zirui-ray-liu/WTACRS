#include "hip/hip_runtime.h"
// our code is based-off the official codes of ActNN and BLPA.
// ActNN: Reducing Training Memory Footprint via 2-Bit Activation Compressed Training.
// Backprop with Approximate Activations for Memory-efficient Network Training.

// Here we summarize the common parts and the difference.
// 1. ActNN and BLPA hack the byte format in a same way to maximize the memory saving. 
//    We just follow their design principles.

// 2. To maximize the memory saving, we must provide a quantized version for 
//    all of the non-linear operations, e.g., ReLU and LeakyReLU. 
//    Please see Appendix H.2 for a detailed explanation.
//    This part is usually got ignored in BLPA and many previous works. We note that although
//    theorectically the mask matrix in ReLU only take one bit per element, however,
//    Pytorch actually cannot go below one byte due to some engineering trade-off.
//    Please see https://github.com/pytorch/pytorch/issues/41571 for a detailed explanation.
//    To maximize the memory saving, we need to hack the byte format for the mask matrix to achieve
//    a space complexity of one-bit per element.
//    We found that ActNN's quantized ReLU is near-optimal. 
//    Here we just utilize their implementation,
//    and extend it for LeakyReLU and ELU, which is commonly used in GAT.

// 3. For the dropout function, similar to the ReLU and LeakyReLU function, we need to provide
//    a quantized version for the Dropout function. We provide our implementation. We note that
//    ActNN also has one quantized dropout function, however, it runs too slow compared to Pytorch's
//    official dropout function and ours. We note that the run time speed of our quantized dropout 
//    is near-optimal.

// 4. We write some glue codes in C++ such that we can insert the quantization & random projection 
//    in the forward pass & backward pass of common OPs. See backward_func.cc and backward_func_cuda_kernel.cu

#include <stdio.h>
#include <torch/extension.h>
// #include <ATen/CUDAGeneratorImpl.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/macros/Macros.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>


#define BLOCK_Y_DIM_MAX ((1l << 16) - 1)

using torch::IntArrayRef;
using torch::Tensor;



/****************************************/
/********** Act Quantized ReLU **********/
/****************************************/
#define ACT_QUANTIZED_RELU_NUM_THREADS 512
// Unpack int32 bit stream to float16/32 data
template <typename scalar_t>
__global__ void act_quantized_relu_forward_kernel(const scalar_t* __restrict__ data,
                                                  int32_t* __restrict__ mask,
                                                  scalar_t* __restrict__ output,
                                                  int64_t N,
                                                  int64_t mask_len) {
  const int64_t id = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t global_offset = (int64_t)blockIdx.x * blockDim.x / (sizeof(int32_t) * 8);
  const int shared_len = ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8);
  __shared__ int mask_shared[ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8)];

  if (threadIdx.x * 2 < shared_len) {
    reinterpret_cast<int2*>(mask_shared)[threadIdx.x] = make_int2(0, 0);
  }

  if (id < N) {
    bool bit = data[id] > 0;
    if (bit) {
      output[id] = data[id];
    } else {
      output[id] = 0.0;
    }

    __syncthreads();
    atomicOr(mask_shared + threadIdx.x % shared_len, bit << (threadIdx.x / shared_len));
    __syncthreads();
  }

  if (threadIdx.x * 2 < shared_len) {
    reinterpret_cast<int2*>(mask)[global_offset / 2 + threadIdx.x] = reinterpret_cast<int2*>(mask_shared)[threadIdx.x];
  }
}

std::pair<Tensor, Tensor> act_quantized_relu_forward_cuda(Tensor data) {
  int64_t n_elements = 1;
  for (size_t i = 0; i < data.dim(); ++i) {
    n_elements *= data.size(i);
  }

  auto options = torch::TensorOptions().dtype(torch::kInt32).device(data.device());
  int64_t mask_len = (n_elements + sizeof(int32_t) * 8 - 1) / (sizeof(int32_t) * 8);
  Tensor mask = torch::empty({mask_len}, options);
  Tensor output = torch::empty_like(data);

  int threads = ACT_QUANTIZED_RELU_NUM_THREADS;
  int blocks = (n_elements + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "act_quantized_relu_forward", ([&] {
    act_quantized_relu_forward_kernel<scalar_t><<<blocks, threads>>>(
      data.data_ptr<scalar_t>(), mask.data_ptr<int32_t>(), output.data_ptr<scalar_t>(),
      n_elements, mask_len);
  }));

  return std::make_pair(output, mask);
}

template <typename scalar_t>
__global__ void act_quantized_relu_backward_kernel(const scalar_t* __restrict__ grad_output,
                                                   int32_t* __restrict__ mask,
                                                   scalar_t* __restrict__ grad_input,
                                                   int N) {
  int64_t id = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t global_offset = (int64_t)blockIdx.x * blockDim.x / (sizeof(int32_t) * 8);
  const int shared_len = ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8);

  if (id < N) {
    bool bit =  (mask[global_offset + threadIdx.x % shared_len] >> (threadIdx.x / shared_len)) & 1;
    if (bit) {
      grad_input[id] = grad_output[id];
    } else {
      grad_input[id] = 0.0;
    }
  }
}


Tensor act_quantized_relu_backward_cuda(Tensor grad_output, Tensor mask) {
  int64_t n_elements = 1;
  for (size_t i = 0; i < grad_output.dim(); ++i) {
    n_elements *= grad_output.size(i);
  }

  int threads = ACT_QUANTIZED_RELU_NUM_THREADS;
  int blocks = (n_elements + threads - 1) / threads;

  Tensor grad_input = torch::empty_like(grad_output);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "act_quantized_relu_backward", ([&] {
      act_quantized_relu_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output.data_ptr<scalar_t>(), mask.data_ptr<int32_t>(), grad_input.data_ptr<scalar_t>(),
        n_elements);
  }));

  return grad_input;
}


/****************************************/
/********** Act Quantized Dropout *******/
/****************************************/
#define ACT_QUANTIZED_DROPOUT_NUM_THREADS 512
#define UNROLL 4
template <typename scalar_t, int ADims, int BDims=ADims>
__global__ void act_quantized_dropout_forward_kernel(at::cuda::detail::TensorInfo<scalar_t, int64_t> a,
                                                  int32_t* __restrict__ mask,
                                                  at::cuda::detail::TensorInfo<scalar_t, int64_t> b,
                                                  std::pair<uint64_t, uint64_t> seeds,
                                                  int64_t N,
                                                  int64_t mask_len,
                                                  float p) {
  const int64_t id = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int shared_len = ACT_QUANTIZED_DROPOUT_NUM_THREADS / (sizeof(int32_t) * 8);
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, id, seeds.second, &state);
  const int64_t rounded_size = ((N - 1)/(blockDim.x * gridDim.x * UNROLL)+1) * blockDim.x * gridDim.x * UNROLL;
  float pinv = 1. / (1. - p);
  __shared__ int mask_shared[shared_len*UNROLL];
  for (int64_t linearIndex = id;
       linearIndex < rounded_size;
       linearIndex += gridDim.x * blockDim.x * UNROLL) {
       const int64_t global_offset = (int64_t)(blockIdx.x * blockDim.x + linearIndex - id) / (sizeof(int32_t) * 8);
       int64_t local_offset = (int64_t)blockDim.x * gridDim.x / (sizeof(int32_t) * 8);
      //hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
       float4 rand = hiprand_uniform4(&state);
       scalar_t src[UNROLL];
       bool inrange[UNROLL] = {0};
       rand.x = rand.x > p;
       rand.y = rand.y > p;
       rand.z = rand.z > p;
       rand.w = rand.w > p;
       if (threadIdx.x * 2 < shared_len) {
        for (int ii = 0; ii < UNROLL; ii++) {
          reinterpret_cast<int2*>(mask_shared)[threadIdx.x+ii*shared_len/2] = make_int2(0, 0);}
       }
       for (int ii = 0; ii < UNROLL; ii++) {
           int64_t li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < N) {
              // Convert `linearIndex` into an offset of `a`
               int64_t aOffset =
                   at::cuda::detail::IndexToOffset<scalar_t, int64_t, ADims>::get(li, a);
               src[ii] = a.data[aOffset];
               inrange[ii] = 1;
           }
       }
       for (int ii = 0; ii < UNROLL; ii++) {
           int64_t li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < N) {
              // Convert `linearIndex` into an offset of `b`
               const int64_t bOffset =
                   at::cuda::detail::IndexToOffset<scalar_t, int64_t, BDims>::get(li, b);
               b.data[bOffset] = src[ii]*(&rand.x)[ii]*pinv;
           }
       }
       __syncthreads();
      for (int ii = 0; ii < UNROLL; ii++) {
        bool bit = (&rand.x)[ii];
        if (inrange[ii]){
          atomicOr(mask_shared+ii*shared_len+threadIdx.x%shared_len, bit << (threadIdx.x/shared_len));}
      }
       __syncthreads();
      
      if (threadIdx.x * 2 < shared_len) {
        for (int ii = 0; ii < UNROLL; ii++){
          if (inrange[ii]){
            reinterpret_cast<int2*>(mask)[threadIdx.x+global_offset/2+ii*local_offset/2] = reinterpret_cast<int2*>(mask_shared)[threadIdx.x+ii*shared_len/2];}
          }
      }
  }
}

std::pair<Tensor, Tensor> act_quantized_dropout_forward_cuda(Tensor data, float p) {
  int64_t n_elements = 1;
  for (size_t i = 0; i < data.dim(); ++i) {
    n_elements *= data.size(i);
  }

  auto options = torch::TensorOptions().dtype(torch::kInt32).device(data.device());
  int64_t mask_len = (n_elements + sizeof(int32_t) * 8 - 1) / (sizeof(int32_t) * 8);
  Tensor mask = torch::empty({mask_len}, options);
  Tensor output = torch::empty_like(data);

  int64_t block_size = ACT_QUANTIZED_DROPOUT_NUM_THREADS;
  unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
  dim3 dim_block(block_size);
  dim3 grid((n_elements + block_size -1)/block_size);
  grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
  int64_t counter_offset = ((n_elements - 1)/(block_size*grid.x*UNROLL)+1)*UNROLL;
  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(counter_offset);
  }
                            
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "act_quantized_dropout_forward", ([&] {
    auto data_info =
      at::cuda::detail::getTensorInfo<scalar_t, int64_t>(data);
    auto output_info =
      at::cuda::detail::getTensorInfo<scalar_t, int64_t>(output);
    data_info.collapseDims();
    output_info.collapseDims();
    act_quantized_dropout_forward_kernel<scalar_t, 1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
      data_info, mask.data_ptr<int32_t>(), output_info, rng_engine_inputs,
      n_elements, mask_len, p);
  }));

  return std::make_pair(output, mask);
}


template <typename scalar_t>
__global__ void act_quantized_dropout_backward_kernel(const scalar_t* __restrict__ grad_output,
                                                   int32_t* __restrict__ mask,
                                                   scalar_t* __restrict__ grad_input,
                                                   int N,
                                                   float p1m) {
  int64_t id = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int shared_len = ACT_QUANTIZED_DROPOUT_NUM_THREADS / (sizeof(int32_t) * 8);

   for (int64_t linearIndex = id;
       linearIndex < N;
       linearIndex += gridDim.x * blockDim.x) {
         const int64_t global_offset = (int64_t)(blockIdx.x * blockDim.x+linearIndex-id) / (sizeof(int32_t) * 8);
         bool bit =  (mask[global_offset + threadIdx.x % shared_len] >> (threadIdx.x / shared_len)) & 1;
         if (bit){
           grad_input[linearIndex] = grad_output[linearIndex] / p1m;
         }else{
           grad_input[linearIndex] = 0.0;
         }
  }
}


Tensor act_quantized_dropout_backward_cuda(Tensor grad_output, Tensor mask, float p1m) {
  int64_t n_elements = 1;
  for (size_t i = 0; i < grad_output.dim(); ++i) {
    n_elements *= grad_output.size(i);
  }

  Tensor grad_input = torch::empty_like(grad_output);
  int64_t block_size = ACT_QUANTIZED_DROPOUT_NUM_THREADS;
  unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
  dim3 dim_block(block_size);
  dim3 grid((n_elements + block_size -1)/block_size);
  grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "act_quantized_dropout_backward", ([&] {
      act_quantized_dropout_backward_kernel<scalar_t><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        grad_output.data_ptr<scalar_t>(), mask.data_ptr<int32_t>(), grad_input.data_ptr<scalar_t>(),
        n_elements, p1m);
  }));
  

  return grad_input;
}